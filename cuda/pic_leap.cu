#include "hip/hip_runtime.h"
#include "pic_cuda.cu"

using namespace std;
using namespace pic_cuda;

#define gpu_error(ans) { gpu_assert((ans), __LINE__); }

inline void gpu_assert(hipError_t code, int line){
  if (code != hipSuccess)
    cerr<<"GPUerror: "<<hipGetErrorString(code)<<" in "<< line<<endl;
}

void initialize_vectors (double *pos_x, double *pos_y, double *vel_x, double *vel_y, double *E_X) {
  int NSP = MAX_SPE;
  for(int i = 0; i < NSP; i++) {
    pos_x[i] = rand() % int(L_MAX_X);
    pos_y[i] = rand() % int(L_MAX_Y);
    vel_x[i] = (rand() % 100) / 2.0;
    vel_y[i] = (rand() % 100) / 2.0;
  }
  for(int i = 0; i < J_X * J_Y; i++) {
    E_X[i] = rand() % 2342;
  }
}

int main() {
  //************************
  // Parámetros del sistema
  //************************

  int le = MAX_SPE, li = MAX_SPE;
  int  total_e_perdidos = 0;
  int  total_i_perdidos = 0;
  double  mv2perdidas = 0;

  double  ND = NE03D * pow(LAMBDA_D,3);                          //Parámetro del plasma
 // FILE    *outEnergia;


  //***************************
  //Constantes de normalización
  //***************************

  //double  X0 = LAMBDA_D;                //Escala de longitud: Longitud de Debye
  double  ni0_3D  =  NI03D * pow(X0, 3);
  double  ne0_3D  =  NE03D * pow(X0, 3);

  int size = MAX_SPE * sizeof(double);
  int size1 = J_X * J_Y * sizeof(double);

  double *pos_e_x, *pos_e_y, *pos_i_x, *pos_i_y, *vel_e_x, *vel_e_y, *vel_i_x, *vel_i_y, *ne, *ni;
  double *phi, *E_X, *E_Y;
  //double  E_i,E_e,E_field,E_total,E_perdida;

  pos_e_x = (double *) malloc(size);
  pos_e_y = (double *) malloc(size);
  pos_i_x = (double *) malloc(size);
  pos_i_y = (double *) malloc(size);

  vel_e_x = (double *) malloc(size);
  vel_e_y = (double *) malloc(size);
  vel_i_x = (double *) malloc(size);
  vel_i_y = (double *) malloc(size);
  ne    = (double *) malloc(size1);
  ni    = (double *) malloc(size1);
  phi   = (double *) malloc(size1);
  E_X   = (double *) malloc(size1);
  E_Y   = (double *) malloc(size1);
  phi   = (double *) malloc(size1);

  //***************************
  // Normalización de variables
  //***************************

  double hx = DELTA_X / X0;                            // Paso espacial
  int max_it = 20;

  double tcon, tscon, telec, tselec, tmot, tsmot;
  tcon = tscon = telec = tselec = tmot = tsmot = 0.0 ;
  clock_t tiempo;
  cout << "start " << endl;
  for(int it  =  0; it <= max_it; it++) {
    cout << it << endl;
    initialize_vectors (pos_e_x, pos_e_y, vel_e_x, vel_e_y, E_X);
    initialize_vectors (pos_i_x, pos_i_y, vel_i_x, vel_i_y, E_Y);
    for(int i = 0; i < J_X * J_Y; i++)
      phi[i] =  rand() % 8234;

    // Calculo de "densidad de carga 2D del plasma"
    tiempo = clock();
    H_Concentration (pos_e_x, pos_e_y, ne, le, hx);// Calcular concentración de superpartículas electrónicas
    gpu_error(hipGetLastError());
    H_Concentration (pos_i_x, pos_i_y, ni, li, hx);// Calcular concentración de superpartículas Iónicas
    gpu_error(hipGetLastError());
    tcon += clock() - tiempo;

    tiempo = clock();
    Concentration (pos_e_x, pos_e_y, ne, le, hx);// Calcular concentración de superpartículas electrónicas
    Concentration (pos_i_x, pos_i_y, ni, li, hx);// Calcular concentración de superpartículas Iónicas
    tscon += clock() - tiempo;

    // Calcular campo eléctrico en puntos de malla
    tiempo = clock();
    H_electric_field(phi, E_X, E_Y, hx);
    gpu_error(hipGetLastError());
    telec += clock() - tiempo;

    tiempo = clock();
    electric_field(phi, E_X, E_Y, hx);
    tselec += clock() - tiempo;

    // Avanzar posiciones de superpartículas electrónicas e Iónicas
    tiempo = clock();
    H_Motion(pos_e_x, pos_e_y, vel_e_x, vel_e_y, le, ELECTRONS, E_X, E_Y, hx, total_e_perdidos, mv2perdidas);//, total_elec_perdidos, total_ion_perdidos, mv2_perdidas);
    gpu_error(hipGetLastError());
    H_Motion(pos_i_x, pos_i_y, vel_i_x, vel_i_y, li, IONS, E_X, E_Y, hx, total_i_perdidos, mv2perdidas);//, total_elec_perdidos, total_ion_perdidos, mv2_perdidas);
    gpu_error(hipGetLastError());
    tmot += clock() - tiempo;

    tiempo = clock();
    Motion(pos_e_x, pos_e_y, vel_e_x, vel_e_y, le, ELECTRONS, E_X, E_Y, hx, total_e_perdidos, mv2perdidas);//, total_elec_perdidos, total_ion_perdidos, mv2_perdidas);
    Motion(pos_i_x, pos_i_y, vel_i_x, vel_i_y, li, IONS, E_X, E_Y, hx, total_i_perdidos, mv2perdidas);//, total_elec_perdidos, total_ion_perdidos, mv2_perdidas);
    tsmot += clock() - tiempo;

  } //Cierre del ciclo principal
  cout << "Concentration\nGPU = " << tcon / CLOCKS_PER_SEC << " sec  CPU = " << tscon / CLOCKS_PER_SEC << endl;
  cout << "Electric field\nGPU = " << telec / CLOCKS_PER_SEC << " sec  CPU = " << tsmot / CLOCKS_PER_SEC << endl;
  cout << "Motion\nGPU = " << tmot / CLOCKS_PER_SEC << " sec CPU = " << tsmot / CLOCKS_PER_SEC << endl;
  free(pos_e_x);
  free(pos_e_y);
  free(pos_i_x);
  free(pos_i_y);
  free(vel_e_x);
  free(vel_e_y);
  free(vel_i_x);
  free(vel_i_y);
  free(ne);
  free(ni);
  free(phi);
  free(E_X);
  free(E_Y);

  return (0);
}// FINAL MAIN

